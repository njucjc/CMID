
#include <hip/hip_runtime.h>

#include <stdio.h>
/*
enum Type {
	NOT_NODE = 0,
	AND_NODE,
	IMPLIES_NODE,
	UNIVERSAL_NODE,
	EXISTENTIAL_NODE,
	BFUNC_NODE,
	EMPTY_NODE,
	SAME,
	SZ_SPD_CLOSE,
	SZ_LOC_CLOSE,
	SZ_LOC_DIST,
	SZ_LOC_DIST_NEQ ,
	SZ_LOC_RANGE,
	OR_NODE
};*/

#define NOT_NODE 0
#define AND_NODE 1
#define IMPLIES_NODE 2
#define UNIVERSAL_NODE 3
#define EXISTENTIAL_NODE 4
#define BFUNC_NODE 5
#define EMPTY_NODE 6
#define SAME 7
#define SZ_SPD_CLOSE 8
#define SZ_LOC_CLOSE 9
#define SZ_LOC_DIST 10
#define SZ_LOC_DIST_NEQ 11
#define SZ_LOC_RANGE 12
#define OR_NODE 13

#define MAX_PARAM_NUM 2
#define MAX_CCT_SIZE 3000000
#define MAX_LINK_SIZE 5000
#define DEBUG

struct Context{
	int id;
	double latitude;
	double longitude;
	double speed;
	int plateNumber;
};

struct Node {
	Node *next;
	Node *tail;
	int params[MAX_PARAM_NUM];
};


__device__ bool truth_values[MAX_CCT_SIZE];
__device__ Node links[MAX_CCT_SIZE];

extern "C"
__device__ bool same(Context c1, Context c2){
	return (c1.plateNumber == c2.plateNumber);
}

extern "C"
__device__ bool sz_spd_close(Context c1, Context c2){
	return ((c1.speed - c2.speed) >= -50.0 && (c1.speed - c2.speed) <= 50.0);
}

extern "C"
__device__ bool sz_loc_close(Context c1, Context c2){
	return ((c1.latitude - c2.latitude) * (c1.latitude - c2.latitude) + (c1.longitude - c2.longitude) * (c1.longitude - c2.longitude)) <= 0.000001;
}

extern "C"
__device__ bool sz_loc_dist(Context c1, Context c2){
	return ((c1.latitude - c2.latitude) * (c1.latitude - c2.latitude) + (c1.longitude - c2.longitude) * (c1.longitude - c2.longitude)) <= 0.000625;
}

extern "C"
__device__ bool sz_loc_dist_neq(Context c1, Context c2){
	double dist = ((c1.latitude - c2.latitude) * (c1.latitude - c2.latitude) + (c1.longitude - c2.longitude) * (c1.longitude - c2.longitude));
	bool result = true;
    if (dist > 0.000625 || dist == 0) {
    	result = false;
    }
    return result;
	//return (dist <= 0.000625) && (dist != 0);
}

extern "C"
__device__ bool sz_loc_range(Context c){
	return c.longitude >= 112.0 && c.longitude <= 116.0 && c.latitude >=20.0 && c.latitude <= 24.0;
}

extern "C"
__device__ void init_node(Node *n){
	n->next = NULL;
	n->tail = n;
	for (int i = 0; i < MAX_PARAM_NUM; i++) {
		n->params[i] = -1;
	}
}

extern "C"
__device__ bool is_null_node(Node *n){
	bool res = true;
	for (int i = 0; i < MAX_PARAM_NUM; i++) {
		res = res && (n->params[i] == -1);
	}
	return res;
}


extern "C"
__device__ void linkHelper(Node *link1, Node *link2) {
	//inital and assumpt that link1 != null, links != null
	if (is_null_node(link1)) {
		for (int i = 0; i < MAX_PARAM_NUM; i++) {
			link1->params[i] = link2->params[i];
		}
		link1->next = NULL;
		link1->tail = link1;

		
		if(link2->next != NULL) {
			link2->next->tail = link2->tail;
		}
		link2 = link2->next;
	}

	if (link2 == NULL) {
		return;
	}

	link1->tail->next = link2;
	link1->tail = link2->tail;
}

extern "C"
__device__ int calc_offset(	int node, int tid, Context *params,
							int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx,
							int *pattern_begin, int *pattern_length, int *pattern,
							double *longitude, double *latitude, double *speed, int *plateNumber, // contexts
							int *branch_size) {

	int offset = branch_size[node];
	int current_node = node;
	int index = 0, tmp = tid;
	while (parent[current_node] != -1) {
		int type = node_type[parent[current_node]];
		if (type == EXISTENTIAL_NODE || type == UNIVERSAL_NODE) {
			int len = pattern_length[pattern_idx[parent[current_node]]];
			int branch_idx = tmp % len;
			tmp /= len;

			params[index].id = pattern[pattern_begin[pattern_idx[parent[current_node]]] + branch_idx];//(pattern + pattern_idx[parent[current_node]] * MAX_PATTERN_SIZE)[(branch_idx + pattern_begin[pattern_idx[parent[current_node]]]) % MAX_PATTERN_SIZE];
			params[index].latitude = latitude[params[index].id];
			params[index].longitude = longitude[params[index].id];
			params[index].speed = speed[params[index].id];
			params[index].plateNumber = plateNumber[params[index].id];

			offset += branch_idx * branch_size[current_node] ;
//			printf("branch_idx = %d, branch_size = %d\n", branch_idx, branch_size[current_node]);
			index++;
		}
		else if (type == AND_NODE || type == IMPLIES_NODE || type == OR_NODE) {
			if (right_child[parent[current_node]] == current_node) {
				offset += branch_size[left_child[parent[current_node]]];
			}
		}
		else {
		    offset += 0;
		}
		current_node = parent[current_node];
	}
	return offset - 1;
}

extern "C"
__global__ void evaluation(int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx, //constraint rule
                          	 int *branch_size, int cunit_begin, int cunit_end,//cunit_end is the root of cunit
                          	 int *pattern_begin, int *pattern_length, int *pattern, //patterns
                          	 double *longitude, double *latitude, double *speed,int *plateNumber,// contexts
                          	 short *truth_value_result,
                          	 int *link_result, int *link_num, int *cur_link_size,
                          	 int last_cunit_root,
                          	 int ccopy_num) {
	
	
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < ccopy_num) {

		Context params[MAX_PARAM_NUM];
		for (int i = 0; i < MAX_PARAM_NUM; i++) {
            params[i].id = -1;
         }
		int ccopy_root_offset = calc_offset(cunit_end, tid, params,
											parent, left_child, right_child, node_type, pattern_idx,
											pattern_begin, pattern_length, pattern,
											longitude, latitude, speed, plateNumber,
											branch_size);

//#ifdef DEBUG
//		printf("root = %d, ccopynum = %d, offset = %d\n",cunit_end, ccopy_num, ccopy_root_offset);
//#endif
		for (int node = cunit_begin; node <= cunit_end; node++) {
			int offset = ccopy_root_offset - (cunit_end - node);
			int type = node_type[node];
			bool value;

			Node* cur_links = &links[offset];
			init_node(cur_links);

			switch(type) {
				case UNIVERSAL_NODE: {
					int step = branch_size[left_child[node]];
					value = true;
					bool first = true;
					for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
						value = value && truth_values[offset - (i * step + 1)];
						if(!truth_values[offset - (i * step + 1)]) {
							if(first) {
								init_node(cur_links);
								first = false;
							}
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
						else if(value) {
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
					}

					break;
				}

				case EXISTENTIAL_NODE: {
					int step = branch_size[left_child[node]];
					value = false;
					bool first = true;
					for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
						value = value || truth_values[offset - (i * step + 1)];
						if(truth_values[offset - (i * step + 1)]) {
							if(first) {
								init_node(cur_links);
								first = false;
							}
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
						else if(!value) {
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
					}
					break;
				}

				case AND_NODE: {
					//right && left
					value = truth_values[offset - 1] && truth_values[offset - (branch_size[right_child[node]] + 1)];

					if (truth_values[offset - 1] == value) {
						linkHelper(cur_links, &(links[offset - 1]));
					}

					if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
						linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}

					break;
				}
				case OR_NODE: {
					//right || left
					value = truth_values[offset - 1] || truth_values[offset - (branch_size[right_child[node]] + 1)];

					if (truth_values[offset - 1] == value) {
						linkHelper(cur_links, &(links[offset - 1]));
					}

					if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
						linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}

					break;
				}

				case IMPLIES_NODE: {
					//!left || right
					value = !truth_values[offset - (branch_size[right_child[node]] + 1)] || truth_values[offset - 1];

					if(value) {
	                   linkHelper(cur_links, &(links[offset - 1]));
	                   linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}
					else {
					   linkHelper(cur_links, &(links[offset - 1]));
					}

					break;
				}

				case NOT_NODE: {
					value = !truth_values[offset - 1];
					linkHelper(cur_links, &(links[offset - 1]));
					break;
				}

				default : { //BFUNC
					switch(type) {
						case SAME: {
							value = same(params[0], params[1]);
							break;
						}

						case SZ_SPD_CLOSE: {
							value = sz_spd_close(params[0], params[1]);
							break;
						}

						case SZ_LOC_CLOSE: {
							value = sz_loc_close(params[0], params[1]);
							break;
						}

						case SZ_LOC_DIST: {
							value = sz_loc_dist(params[0], params[1]);
							break;
						}

						case SZ_LOC_DIST_NEQ: {
							value = sz_loc_dist_neq(params[0], params[1]);
							break;
						}

						case SZ_LOC_RANGE: {
							value = sz_loc_range(params[0]);
							break;
						}
					}

			
					for (int i = 0; i < MAX_PARAM_NUM; i++) {
						cur_links->params[i] = params[i].id;
					}
					break;
				}

				
			}

			truth_values[offset] = value;
		}

		if (last_cunit_root == cunit_end ) {
		    *truth_value_result = truth_values[ccopy_root_offset];
		    if(!truth_values[ccopy_root_offset]) {
            
         		int len = 0;
                for(Node *head = &links[ccopy_root_offset]; head != NULL; head = head ->next) {
                
                	if(len < MAX_LINK_SIZE) {
	                	for(int j = 0; j < MAX_PARAM_NUM; j++) {
	                         link_result[MAX_PARAM_NUM * len + j] = head->params[j];
	                    }
                	}

                    len++;
                }
                
                *cur_link_size = len;
                *link_num = len > MAX_LINK_SIZE ? MAX_LINK_SIZE : len;
         	}
        }
	}

 }

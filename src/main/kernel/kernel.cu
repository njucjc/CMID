
#include <hip/hip_runtime.h>

#include <stdio.h>

enum Type {
	NOT_NODE = 0,
	AND_NODE,
	IMPLIES_NODE,
	UNIVERSAL_NODE,
	EXISTENTIAL_NODE,
	BFUNC_NODE,
	EMPTY_NODE,
	SAME,
	SZ_SPD_CLOSE,
	SZ_LOC_CLOSE,
	SZ_LOC_DIST,
	SZ_LOC_DIST_NEQ ,
	SZ_LOC_RANGE,
	OR_NODE
};

#define MAX_PARAM_NUM 2
#define MAX_PATTERN_SIZE 500
#define MAX_LINK_SIZE 40
#define DEBUG

struct Context{
	int id;
	double latitude;
	double longitude;
	double speed;
};

struct Links {
	int length;
	int link_pool[MAX_LINK_SIZE][MAX_PARAM_NUM];
};

extern "C"
__device__ bool same(Context c1, Context c2){
	return (c1.id == c2.id);
}

extern "C"
__device__ bool sz_spd_close(Context c1, Context c2){
	return ((c1.speed - c2.speed) >= -50.0 && (c1.speed - c2.speed) <= 50.0);
}

extern "C"
__device__ bool sz_loc_close(Context c1, Context c2){
	return ((c1.latitude - c2.latitude) * (c1.latitude - c2.latitude) + (c1.longitude - c2.longitude) * (c1.longitude - c2.longitude)) <= 0.000001;
}

extern "C"
__device__ bool sz_loc_dist(Context c1, Context c2){
	return ((c1.latitude - c2.latitude) * (c1.latitude - c2.latitude) + (c1.longitude - c2.longitude) * (c1.longitude - c2.longitude)) <= 0.000625;
}

extern "C"
__device__ bool sz_loc_dist_neq(Context c1, Context c2){
	double dist = ((c1.latitude - c2.latitude) * (c1.latitude - c2.latitude) + (c1.longitude - c2.longitude) * (c1.longitude - c2.longitude));
	bool result = true;
    if (dist > 0.000625 || dist == 0) {
    	result = false;
    }
    return result;
	//return (dist <= 0.000625) && (dist != 0);
}

extern "C"
__device__ bool sz_loc_range(Context c){
	return c.longitude >= 112.0 && c.longitude <= 116.0 && c.latitude >=20.0 && c.latitude <= 24.0;
}

extern "C"
__device__ void linkHelper(Links *left, Links *right) {
	int left_len = left->length;
	int right_len = right->length;


	for (int i = 0; i < right_len; i++) {
		int j;
		for (j = 0; j < left_len; j++) {
			if (right->link_pool[i][0] == left->link_pool[j][0] && right->link_pool[i][1] == left->link_pool[j][1]) {
				break;
			}
		}
		if (j == left_len) {
			left->link_pool[left->length][0] = right->link_pool[i][0];
			left->link_pool[left->length][1] = right->link_pool[i][1];
			left->length = (left->length + 1) % MAX_LINK_SIZE;
		}

	}
}

extern "C"
__device__ int calc_offset(	int node, int tid, Context *params,
							int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx,
							int *pattern_begin, int *pattern_length, int *pattern,
							double *longitude, double *latitude, double *speed, // contexts
							int *branch_size) {

	int offset = branch_size[node];
	int current_node = node;
	int index = 0, tmp = tid;
	while (parent[current_node] != -1) {
		int type = node_type[parent[current_node]];
		if (type == Type::EXISTENTIAL_NODE || type == Type::UNIVERSAL_NODE) {
			int len = pattern_length[pattern_idx[parent[current_node]]];
			int branch_idx = tmp % len;
			tmp /= len;

			params[index].id = (pattern + pattern_idx[parent[current_node]] * MAX_PATTERN_SIZE)[(branch_idx + pattern_begin[pattern_idx[parent[current_node]]]) % MAX_PATTERN_SIZE];
			params[index].latitude = latitude[params[index].id];
			params[index].longitude = longitude[params[index].id];
			params[index].speed = speed[params[index].id];

			offset += branch_idx * branch_size[current_node] ;
//			printf("branch_idx = %d, branch_size = %d\n", branch_idx, branch_size[current_node]);
			index++;
		}
		else if (type == Type::AND_NODE || type == Type::IMPLIES_NODE || type == Type::OR_NODE) {
			if (right_child[parent[current_node]] == current_node) {
				offset += branch_size[left_child[parent[current_node]]];
			}
		}
		else {
		    offset += 0;
		}
		current_node = parent[current_node];
	}
	return offset - 1;
}



extern "C"
__global__ void gen_truth_value(int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx, //constraint rule
								int *branch_size, int cunit_begin, int cunit_end,//cunit_end is the root of cunit
								int *pattern_begin, int *pattern_length, int *pattern, //patterns
								double *longitude, double *latitude, double *speed, // contexts
								short *truth_values, int ccopy_num) {
	
	
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < ccopy_num) {

		Context params[MAX_PARAM_NUM];
		int ccopy_root_offset = calc_offset(cunit_end, tid, params,
											parent, left_child, right_child, node_type, pattern_idx,
											pattern_begin, pattern_length, pattern,
											longitude, latitude, speed,
											branch_size);
//#ifdef DEBUG
//		printf("root = %d, ccopynum = %d, offset = %d\n",cunit_end, ccopy_num, ccopy_root_offset);
//#endif
		for (int node = cunit_begin; node <= cunit_end; node++) {
			int offset = ccopy_root_offset - (cunit_end - node);
			int type = node_type[node];
			bool value;
			if(type == Type::UNIVERSAL_NODE) {
				int step = branch_size[left_child[node]];
				value = true;
				for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
					value = value && truth_values[offset - (i * step + 1)];
				}
			}
			else if (type == Type::EXISTENTIAL_NODE) {
				int step = branch_size[left_child[node]];
				value = false;
				for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
					value = value || truth_values[offset - (i * step + 1)];
				}
			}
			else if (type == Type::AND_NODE) {
				//right && left
				value = truth_values[offset - 1] && truth_values[offset - (branch_size[right_child[node]] + 1)];
			}
			else if (type == Type::OR_NODE) {
				//right || left
				value = truth_values[offset - 1] || truth_values[offset - (branch_size[right_child[node]] + 1)];
			}
			else if (type == Type::IMPLIES_NODE) {
				//!left || right
				value = !truth_values[offset - (branch_size[right_child[node]] + 1)] || truth_values[offset - 1];
			}
			else if (type == Type::NOT_NODE) {
				value = !truth_values[offset - 1];
			}
			else if (type == Type::SAME) {
				value = same(params[0], params[1]);
			}
			else if (type == Type::SZ_SPD_CLOSE) {
				value = sz_spd_close(params[0], params[1]);
			}
			else if (type == Type::SZ_LOC_CLOSE) {
				value = sz_loc_close(params[0], params[1]);
			}
			else if (type == Type::SZ_LOC_DIST) {
				value = sz_loc_dist(params[0], params[1]);
			}
			else if (type == Type::SZ_LOC_DIST_NEQ) {
				value = sz_loc_dist_neq(params[0], params[1]);
			}
			else if (type == Type::SZ_LOC_RANGE) {
				value = sz_loc_range(params[0]);
			}
			truth_values[offset] = value;
		}
	}

 }

 extern "C"
 __global__ void gen_links(int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx, //constraint rule
	 int *branch_size, int cunit_begin, int cunit_end,//cunit_end is the root of cunit
	 int *pattern_begin, int *pattern_length, int *pattern, //patterns
	 double *longitude, double *latitude, double *speed, // contexts
	 short *truth_values,
	 Links *links, int *link_result, int *link_num,
	 int last_cunit_root,
	 int ccopy_num) {

	 int tid = threadIdx.x + blockDim.x * blockIdx.x;
	 if(tid < ccopy_num) {
		 Context params[MAX_PARAM_NUM];

		 for (int i = 0; i < MAX_PARAM_NUM; i++) {
			 params[i].id = -1;
		 }

		 int ccopy_root_offset = calc_offset(cunit_end, tid, params,
			 parent, left_child, right_child, node_type, pattern_idx,
			 pattern_begin, pattern_length, pattern,
			 longitude, latitude, speed,
			 branch_size);

		 for (int node = cunit_begin; node <= cunit_end; node++) {
			 int offset = ccopy_root_offset - (cunit_end - node);
			 int type = node_type[node];
			 bool value = truth_values[offset];
			 Links* cur_links = &links[offset];
			 cur_links->length = 0;
			 if (type == Type::UNIVERSAL_NODE || type == Type::EXISTENTIAL_NODE) {
				 int step = branch_size[left_child[node]];
				 for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
					 if (truth_values[offset - (i * step + 1)] == value) {
						 linkHelper(cur_links, &links[offset - (i * step + 1)]);
					 }
				 }
//				 printf("length = %d\n", cur_links->length);
//				 for(int i = 0; i < cur_links->length; i++) {
//				    printf("%d %d\n", cur_links->link_pool[i][0], cur_links->link_pool[i][1]);
//				 }
			 }
			 else if (type == Type::AND_NODE || type == Type::OR_NODE) {
				 if (truth_values[offset - 1] == value) {
					 linkHelper(cur_links, &links[offset - 1]);
				 }

				 if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
					 linkHelper(cur_links, &links[offset - (branch_size[right_child[node]] + 1)]);
				 }
			 }
			 else if (type == Type::IMPLIES_NODE) {
				 //!left || right
				 bool left = truth_values[offset - (branch_size[right_child[node]] + 1)];
				 bool right = truth_values[offset - 1];

				 if ((!left && right) || left && !right) {
					 linkHelper(cur_links, &links[offset - 1]);
					 linkHelper(cur_links, &links[offset - (branch_size[right_child[node]] + 1)]);
				 }
				 else if(left && right){
					 linkHelper(cur_links, &links[offset - 1]);
				 }
				 else if (left && right) {
					 linkHelper(cur_links, &links[offset - (branch_size[right_child[node]] + 1)]);
				 }
			 }
			 else if (type == Type::NOT_NODE) {
				 linkHelper(cur_links, &links[offset - 1]);
			 }
			 else if (type == Type::SAME
				 || type == Type::SZ_SPD_CLOSE
				 || type == Type::SZ_LOC_CLOSE
				 || type == Type::SZ_LOC_DIST
				 || type == Type::SZ_LOC_DIST_NEQ
				 || type == Type::SZ_LOC_RANGE) {
				 cur_links->length = 1;
				 for (int i = 0; i < MAX_PARAM_NUM; i++) {
					 cur_links->link_pool[0][i] = params[i].id;
				 }
			 }

		 }

		 if (last_cunit_root == cunit_end) {
         	*link_num = links[ccopy_root_offset].length;
         		for (int i = 0; i < *link_num; i++) {
         			link_result[i] = links[ccopy_root_offset].link_pool[i][0];
         			link_result[i + 1] = links[ccopy_root_offset].link_pool[i][1];
         		}
          }
	 }

  }

extern "C"
__global__ void update_pattern(int op, int pattern_idx,
							   int *pattern_begin, int *pattern_length, int *pattern,
							   int id) {
	if (op == 0) { //-
		pattern_begin[pattern_idx] = (pattern_begin[pattern_idx] + 1) % MAX_PATTERN_SIZE;
		pattern_length[pattern_idx]--;
	}
	else if (op == 1) {//+
		(pattern + pattern_idx * MAX_PATTERN_SIZE)[(pattern_begin[pattern_idx] + pattern_length[pattern_idx]) % MAX_PATTERN_SIZE] = id;
		pattern_length[pattern_idx]++;

	}

}
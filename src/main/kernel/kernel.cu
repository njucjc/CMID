
#include <hip/hip_runtime.h>

#include <stdio.h>

enum Type {
	NOT_NODE = 0,
	AND_NODE,
	IMPLIES_NODE,
	UNIVERSAL_NODE,
	EXISTENTIAL_NODE,
	BFUNC_NODE,
	EMPTY_NODE,
	SAME,
	SZ_SPD_CLOSE,
	SZ_LOC_CLOSE,
	SZ_LOC_DIST,
	SZ_LOC_DIST_NEQ ,
	SZ_LOC_RANGE,
	OR_NODE
};

#define MAX_PARAM_NUM 2

__device__ double distance(double latitude1, double latitude2, double longitude1, double longitude2)
{
	return ((latitude1 - latitude2) * (latitude1 - latitude2) + (longitude1 - longitude2) * (longitude1 - longitude2));
}


__device__ bool same(int id1, int id2)
{
	return id1 == id2;
}

__device__ bool sz_spd_close(double speed1, double speed2) 
{
	return ((speed1 - speed2) >= -50.0 && (speed1 - speed2) <= 50.0);
}

__device__ bool sz_loc_close(double latitude1, double latitude2, double longitude1, double longitude2) 
{
	return distance(latitude1, latitude2, longitude1, longitude2) <= 0.000001;
}

__device__ bool sz_loc_dist(double latitude1, double latitude2, double longitude1, double longitude2) 
{
	return  distance(latitude1, latitude2, longitude1, longitude2) <= 0.000625;
}

__device__ bool sz_loc_dist_neq(double latitude1, double latitude2, double longitude1, double longitude2) 
{
	double dist = distance(latitude1, latitude2, longitude1, longitude2);
	bool result = true;
	if (dist > 0.000625 || dist == 0) {
		result = false;
	}
	return result;
	//return (dist <= 0.000625) && (dist != 0);
}

__device__ bool sz_loc_range(double latitude, double longitude)
{
	return longitude >= 112.0 && longitude <= 116.0 && latitude >= 20.0 && latitude <= 24.0;
}

__device__ int calc_offset(	int node, int tid, int *branch_idx,
							int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx,
							int *pattern_length,
							int *branch_size)
{
	int offset = 0;
	int current_node = node;
	int index = 0, tmp = tid;
	while (parent[current_node] != -1) {
		int type = node_type[parent[current_node]];
		if (type == Type::EXISTENTIAL_NODE || type == Type::UNIVERSAL_NODE) {
			int len = pattern_length[pattern_idx[parent[current_node]]];
			branch_idx[index] = tmp % len;
			tmp /= len;
			offset += (branch_idx[index] + 1) * branch_size[current_node];
			index++;
		}
		else if (type == Type::AND_NODE || type == Type::IMPLIES_NODE || type == Type::OR_NODE) {
			if (right_child[parent[current_node]] == current_node) {
				offset += 2 * branch_size[current_node];
			}
			else {
				offset += branch_size[current_node];
			}
		}
		else {
			offset += branch_size[current_node];
		}
		current_node = parent[current_node];
	}
	return offset == 0 ? 0 : offset - 1;
}



extern "C"
__global__ void gen_truth_value(int *parent, int *left_child, int *right_hild, int *node_type, int *pattern_idx, //constraint rule 
								int *branch_size, int cunit_begin, int cunit_end,//cunit_end is the root of cunit
								int pattern_num, int *pattern_begin, int *pattern_length, int *pattern, //patterns
								double *longitude, double *latitude, double *speed, // contexts
								int *truth_values)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int branch_idx[MAX_PARAM_NUM];
	int ccopy_root_offset = calc_offset(cunit_end, tid, branch_idx, parent, left_child, right_hild, node_type, pattern_idx, pattern_length, branch_size);

	for (int node = cunit_begin; node <= cunit_end; node++) {
		int offset = ccopy_root_offset - (cunit_end - node);

	}

 }

 extern "C"
 __global__ void gen_links()
 {

  }

extern "C"
__global__ void update_pattern()
{

}
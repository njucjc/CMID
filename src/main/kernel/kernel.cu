
#include <hip/hip_runtime.h>

#include <stdio.h>

#define NOT_NODE 0
#define AND_NODE 1
#define IMPLIES_NODE 2
#define UNIVERSAL_NODE 3
#define EXISTENTIAL_NODE 4
#define BFUNC_NODE 5
#define EMPTY_NODE 6

#define ELECTRIC_RANGE 13
#define VOLTAGE_RANGE 14
#define ACC_RANG 15
#define ACC_RATE_RANG 16
#define ALL_IN_BRAKE_STATE 17
#define ALL_IN_TRACTION_STATE 18
#define NOT_TRANS_TO_BRAKE 19
#define NOT_TRANS_TO_TRACTION 20
#define IN_BRAKE_STATE 21
#define IN_TRACTION_STATE 22
#define OR_NODE 23

#define MAX_PARAM_NUM 2
#define MAX_CCT_SIZE 3000000
#define MAX_LINK_SIZE 5000
#define DEBUG


#define STOP 0
#define START 1
#define TRACTION 2
#define COAST 3
#define BRAKE 4

struct Context{
	int id;
	double u;
	double i;
	double p;
	double v;
	double a;
	int status;
};

struct Node {
	Node *next;
	Node *tail;
	int params[MAX_PARAM_NUM];
};


__device__ bool truth_values[MAX_CCT_SIZE];
__device__ Node links[MAX_CCT_SIZE];


extern "C"
__device__ double my_abs(double num){
	if (num < 0) {
		return -num;
	}
	else {
		return num;
	}
}

extern "C"
__device__ int now(Context c1, Context c2, int diff){
	if (c1.id - c2.id == diff) {
		return 2;
	}
	else if (c2.id - c1.id == diff) {
		return 1;
	}
	else {
		return 0;
	}
}

extern "C"
__device__ int next(Context c1, Context c2, int diff){
	if (c1.id - c2.id == diff) {
		return 1;
	}
	else if (c2.id - c1.id == diff) {
		return 2;
	}
	else {
		return 0;
	}
}

extern "C"
__device__ bool electric_range(Context c){
	if (c.status != TRACTION && c.status != BRAKE) {
		return true;
	}
	return abs(c.i) <= 740.0;
}

extern "C"
__device__ bool voltage_range(Context c){
	return abs(c.u) >= 1450.0 && abs(c.u) <= 1800.0;
}

extern "C"
__device__ bool acc_range(Context c1, Context c2){
	bool res = true;
	double t = 5.0;
	double v = abs(c1.v - c2.v);

	if (now(c1, c2, 50) != 0) {
		res = (v / t) <= 1.0;
	}
	return res;
}

extern "C"
__device__ bool acc_rate_range(Context c1, Context c2){
	bool res = true;
	double t = 5.0;
	double a = abs(c1.a - c2.a);

	if (now(c1, c2, 50) != 0) {
		res = (a / t) <= 1.5;
	}
	return res;
}

extern "C"
__device__ bool not_trans_to_brake(Context c1, Context c2){
	int no = next(c1, c2, 1);
	if (no == 1) {
		return c1.status != BRAKE;
	}
	else if (no == 2) {
		return c2.status != BRAKE;
	}
	else {
		return false;
	}
}

extern "C"
__device__ bool not_trans_to_traction(Context c1, Context c2){
	int no = next(c1, c2, 1);
	if (no == 1) {
		return c1.status != TRACTION;
	}
	else if (no == 2) {
		return c2.status != TRACTION;
	}
	else {
		return false;
	}
}

extern "C"
__device__ bool in_brake_state(Context c1, Context c2){
	int no = now(c1, c2, 1);
	if (no == 1) {
		return c1.status == BRAKE;
	}
	else if (no == 2) {
		return c2.status == BRAKE;
	}
	else {
		return false;
	}
}

extern "C"
__device__ bool in_traction_state(Context c1, Context c2){
	int no = now(c1, c2, 1);
	if (no == 1) {
		return c1.status == TRACTION;
	}
	else if (no == 2) {
		return c2.status == TRACTION;
	}
	else {
		return false;
	}
}


extern "C"
__device__ bool all_in_brake_state(Context c1, Context c2){
	if (c1.status == BRAKE && c2.status == BRAKE) {
		return now(c1, c2, 50) != 0;
	}
	return false;
}

extern "C"
__device__ bool all_in_traction_state(Context c1, Context c2){
	if (c1.status == TRACTION && c2.status == TRACTION) {
		return now(c1, c2, 50) != 0;
	}
	return false;
}

extern "C"
__device__ void init_node(Node *n){
	n->next = NULL;
	n->tail = n;
	for (int i = 0; i < MAX_PARAM_NUM; i++) {
		n->params[i] = -1;
	}
}

extern "C"
__device__ bool is_null_node(Node *n){
	bool res = true;
	for (int i = 0; i < MAX_PARAM_NUM; i++) {
		res = res && (n->params[i] == -1);
	}
	return res;
}


extern "C"
__device__ void linkHelper(Node *link1, Node *link2) {
	//inital and assumpt that link1 != null, links != null
	if (is_null_node(link1)) {
		for (int i = 0; i < MAX_PARAM_NUM; i++) {
			link1->params[i] = link2->params[i];
		}
		link1->next = NULL;
		link1->tail = link1;

		
		if(link2->next != NULL) {
			link2->next->tail = link2->tail;
		}
		link2 = link2->next;
	}

	if (link2 == NULL) {
		return;
	}

	link1->tail->next = link2;
	link1->tail = link2->tail;
}

extern "C"
__device__ int calc_offset(	int node, int tid, Context *params,
							int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx,
							int *pattern_begin, int *pattern_length, int *pattern,
							double *u_u, double *i_i, double *v_v, double *p_p, double *a_a, int *status, // contexts
							int *branch_size) {

	int offset = branch_size[node];
	int current_node = node;
	int index = 0, tmp = tid;
	while (parent[current_node] != -1) {
		int type = node_type[parent[current_node]];
		if (type == EXISTENTIAL_NODE || type == UNIVERSAL_NODE) {
			int len = pattern_length[pattern_idx[parent[current_node]]];
			int branch_idx = tmp % len;
			tmp /= len;

			params[index].id = pattern[pattern_begin[pattern_idx[parent[current_node]]] + branch_idx];//(pattern + pattern_idx[parent[current_node]] * MAX_PATTERN_SIZE)[(branch_idx + pattern_begin[pattern_idx[parent[current_node]]]) % MAX_PATTERN_SIZE];
			params[index].u = u_u[params[index].id];
			params[index].i = i_i[params[index].id];
			params[index].v = v_v[params[index].id];
			params[index].p = p_p[params[index].id];
			params[index].a = a_a[params[index].id];
			params[index].status = status[params[index].id];

			offset += branch_idx * branch_size[current_node] ;
//			printf("branch_idx = %d, branch_size = %d\n", branch_idx, branch_size[current_node]);
			index++;
		}
		else if (type == AND_NODE || type == IMPLIES_NODE || type == OR_NODE) {
			if (right_child[parent[current_node]] == current_node) {
				offset += branch_size[left_child[parent[current_node]]];
			}
		}
		else {
		    offset += 0;
		}
		current_node = parent[current_node];
	}
	return offset - 1;
}

extern "C"
__global__ void evaluation(int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx, //constraint rule
                          	 int *branch_size, int cunit_begin, int cunit_end,//cunit_end is the root of cunit
                          	 int *pattern_begin, int *pattern_length, int *pattern, //patterns
                          	 double *u_u, double *i_i, double *v_v, double *p_p, double *a_a, int *status,// contexts
                          	 short *truth_value_result,
                          	 int *link_result, int *link_num, int *cur_link_size,
                          	 int last_cunit_root,
                          	 int ccopy_num) {
	
	
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < ccopy_num) {

		Context params[MAX_PARAM_NUM];
		for (int i = 0; i < MAX_PARAM_NUM; i++) {
            params[i].id = -1;
         }
		int ccopy_root_offset = calc_offset(cunit_end, tid, params,
											parent, left_child, right_child, node_type, pattern_idx,
											pattern_begin, pattern_length, pattern,
											u_u, i_i, v_v, p_p, a_a, status,
											branch_size);

//#ifdef DEBUG
//		printf("root = %d, ccopynum = %d, offset = %d\n",cunit_end, ccopy_num, ccopy_root_offset);
//#endif
		for (int node = cunit_begin; node <= cunit_end; node++) {
			int offset = ccopy_root_offset - (cunit_end - node);
			int type = node_type[node];
			bool value;

			Node* cur_links = &links[offset];
			init_node(cur_links);

			switch(type) {
				case UNIVERSAL_NODE: {
					int step = branch_size[left_child[node]];
					value = true;
					bool first = true;
					for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
						value = value && truth_values[offset - (i * step + 1)];
						if(!truth_values[offset - (i * step + 1)]) {
							if(first) {
								init_node(cur_links);
								first = false;
							}
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
						else if(value) {
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
					}

					break;
				}

				case EXISTENTIAL_NODE: {
					int step = branch_size[left_child[node]];
					value = false;
					bool first = true;
					for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
						value = value || truth_values[offset - (i * step + 1)];
						if(truth_values[offset - (i * step + 1)]) {
							if(first) {
								init_node(cur_links);
								first = false;
							}
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
						else if(!value) {
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
					}
					break;
				}

				case AND_NODE: {
					//right && left
					value = truth_values[offset - 1] && truth_values[offset - (branch_size[right_child[node]] + 1)];

					if (truth_values[offset - 1] == value) {
						linkHelper(cur_links, &(links[offset - 1]));
					}

					if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
						linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}

					break;
				}
				case OR_NODE: {
					//right || left
					value = truth_values[offset - 1] || truth_values[offset - (branch_size[right_child[node]] + 1)];

					if (truth_values[offset - 1] == value) {
						linkHelper(cur_links, &(links[offset - 1]));
					}

					if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
						linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}

					break;
				}

				case IMPLIES_NODE: {
					//!left || right
					value = !truth_values[offset - (branch_size[right_child[node]] + 1)] || truth_values[offset - 1];

					if(value) {
	                   linkHelper(cur_links, &(links[offset - 1]));
	                   linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}
					else {
					   linkHelper(cur_links, &(links[offset - 1]));
					}

					break;
				}

				case NOT_NODE: {
					value = !truth_values[offset - 1];
					linkHelper(cur_links, &(links[offset - 1]));
					break;
				}

				default : { //BFUNC
					switch(type) {
						case ELECTRIC_RANGE: {
							value = electric_range(params[0]);
							break;
						}

						case VOLTAGE_RANGE: {
							value = voltage_range(params[0]);
							break;
						}

						case ACC_RANG: {
							value = acc_range(params[0], params[1]);
							break;
						}

						case ACC_RATE_RANG: {
							value = acc_rate_range(params[0], params[1]);
							break;
						}

						case ALL_IN_BRAKE_STATE: {
							value = all_in_brake_state(params[0], params[1]);
							break;
						}

						case ALL_IN_TRACTION_STATE: {
							value = all_in_traction_state(params[0], params[1]);
							break;
						}

						case NOT_TRANS_TO_BRAKE: {
							value = not_trans_to_brake(params[0], params[1]);
							break;
						}

						case NOT_TRANS_TO_TRACTION: {
							value = not_trans_to_traction(params[0], params[1]);
							break;
						}

						case IN_BRAKE_STATE: {
							value = in_brake_state(params[0], params[1]);
							break;
						}

						case IN_TRACTION_STATE: {
							value = in_traction_state(params[0], params[1]);
							break;
						}

					}

			
					for (int i = 0; i < MAX_PARAM_NUM; i++) {
						cur_links->params[i] = params[i].id;
					}
					break;
				}

				
			}

			truth_values[offset] = value;
		}

		if (last_cunit_root == cunit_end ) {
		    *truth_value_result = truth_values[ccopy_root_offset];
		    if(!truth_values[ccopy_root_offset]) {
            
         		int len = 0;
                for(Node *head = &links[ccopy_root_offset]; head != NULL; head = head ->next) {
                
                	if(len < MAX_LINK_SIZE) {
	                	for(int j = 0; j < MAX_PARAM_NUM; j++) {
	                         link_result[MAX_PARAM_NUM * len + j] = head->params[j];
	                    }
                	}

                    len++;
                }
                
                *cur_link_size = len;
                *link_num = len > MAX_LINK_SIZE ? MAX_LINK_SIZE : len;
         	}
        }
	}

 }

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
/*
enum Type {
	NOT_NODE = 0,
	AND_NODE,
	IMPLIES_NODE,
	UNIVERSAL_NODE,
	EXISTENTIAL_NODE,
	BFUNC_NODE,
	EMPTY_NODE,
	SAME,
	SZ_SPD_CLOSE,
	SZ_LOC_CLOSE,
	SZ_LOC_DIST,
	SZ_LOC_DIST_NEQ ,
	SZ_LOC_RANGE,
	OR_NODE
};*/

#define NOT_NODE 0
#define AND_NODE 1
#define IMPLIES_NODE 2
#define UNIVERSAL_NODE 3
#define EXISTENTIAL_NODE 4
#define BFUNC_NODE 5
#define EMPTY_NODE 6
#define OR_NODE 7

#define BEFORE 8
#define GATE 9
#define EQUAL 10
#define CONN 11
#define OPPO 12
#define NEXT 13

#define MAX_PARAM_NUM 4
#define MAX_CCT_SIZE 3000000
#define MAX_LINK_SIZE 5000

#define GRAPH_NODE_NUM 2000
#define MAX_NEI_NUM 5

#define DEBUG

struct Context{
	int id;
	int code;
	int type;
};

struct Node {
	Node *next;
	Node *tail;
	int params[MAX_PARAM_NUM];
};


__device__ bool truth_values[MAX_CCT_SIZE];
__device__ Node links[MAX_CCT_SIZE];

extern "C"
__device__ bool before(Context c[], int len){
	int tmp = -1;
	for (int i = 0; i < len; i++) {
		if (tmp >= c[i]) return false;
		tmp = c[i];
	}
}

extern "C"
__device__ bool gate(Context c1, Context c2){
	return c1.type == 3 || c2.type == 3;
}

extern "C"
__device__ bool equal(Context c1, Context c2){
	return c1.code == c2.code;
}

extern "C"
__device__ bool conn(int *graph, Context c1, Context c2, int k){
	return has_path(graph, c1.code, c2.code, k);
}

extern "C"
__device__ bool oppo(int *oppo_table, Context c1, Context c2){
	return oppo_table[c1.code] == c2.code;
}

extern "C"
__device__ bool oppo(Context c1, Context c2){
	return c2.id - c1.id == 1;
}

extern "C"
__device__ bool has_path(int *graph, int v, int w, int k){
	bool visited[GRAPH_NODE_NUM];
	for (int i = 0; i < GRAPH_NODE_NUM; i++) {
		visited[i] = false;
	}

	return has_path_k(graph, visited, v, w, k);
}

extern "C"
__device__ bool has_path_k(int *graph, int visited[], int v, int w, int k){
	visited[v] = true;
	if (v == w && k == 0) {
		return true;
	}
	else if (k > 0) {
		int offset = v * MAX_NEI_NUM;
		for (int i = 0; i < MAX_NEI_NUM; i++) {
			if (graph[offset + i] != -1) {
				if (!visited[graph[offset + i]] && has_path_k(graph, visited, graph[offset + i], w, k -1)) return true;
				visited[graph[offset + i]] = false;
			}
			else {
				break;
			}
		}
	}

	return false;
}

extern "C"
__device__ void init_node(Node *n){
	n->next = NULL;
	n->tail = n;
	for (int i = 0; i < MAX_PARAM_NUM; i++) {
		n->params[i] = -1;
	}
}

extern "C"
__device__ bool is_null_node(Node *n){
	bool res = true;
	for (int i = 0; i < MAX_PARAM_NUM; i++) {
		res = res && (n->params[i] == -1);
	}
	return res;
}


extern "C"
__device__ void linkHelper(Node *link1, Node *link2) {
	//inital and assumpt that link1 != null, links != null
	if (is_null_node(link1)) {
		for (int i = 0; i < MAX_PARAM_NUM; i++) {
			link1->params[i] = link2->params[i];
		}
		link1->next = NULL;
		link1->tail = link1;

		
		if(link2->next != NULL) {
			link2->next->tail = link2->tail;
		}
		link2 = link2->next;
	}

	if (link2 == NULL) {
		return;
	}

	link1->tail->next = link2;
	link1->tail = link2->tail;
}

extern "C"
__device__ int calc_offset(	int node, int tid, Context *params,
							int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx,
							int *pattern_begin, int *pattern_length, int *pattern,
							int *codes, int *types, // contexts
							int *branch_size) {

	int offset = branch_size[node];
	int current_node = node;
	int index = 0, tmp = tid;
	while (parent[current_node] != -1) {
		int type = node_type[parent[current_node]];
		if (type == EXISTENTIAL_NODE || type == UNIVERSAL_NODE) {
			int len = pattern_length[pattern_idx[parent[current_node]]];
			int branch_idx = tmp % len;
			tmp /= len;

			params[index].id = pattern[pattern_begin[pattern_idx[parent[current_node]]] + branch_idx];//(pattern + pattern_idx[parent[current_node]] * MAX_PATTERN_SIZE)[(branch_idx + pattern_begin[pattern_idx[parent[current_node]]]) % MAX_PATTERN_SIZE];
			params[index].code = codes[params[index].id];
			params[index].type = types[params[index].id];

			offset += branch_idx * branch_size[current_node] ;
//			printf("branch_idx = %d, branch_size = %d\n", branch_idx, branch_size[current_node]);
			index++;
		}
		else if (type == AND_NODE || type == IMPLIES_NODE || type == OR_NODE) {
			if (right_child[parent[current_node]] == current_node) {
				offset += branch_size[left_child[parent[current_node]]];
			}
		}
		else {
		    offset += 0;
		}
		current_node = parent[current_node];
	}
	return offset - 1;
}

extern "C"
__device__ void reorder_params(int *oppo_table, int *params_order, Context params[], Context ordered_params[]) {
	int len = params_order[0];
	for (int i = 1; i <= len; i++) {
		if (params_order[i] == 0) continue;
		else if (params_order[i] < 0){
			ordered_params[-params_order[i]] = params[i-1]
			ordered_params[-params_order[i]].code = oppo_table[ordered_params[-params_order[i]].code];
		}
		else {
			ordered_params[params_order[i]] = params[i-1];
		}

	}
}

extern "C"
__global__ void evaluation(int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx, //constraint rule
                          	 int *branch_size, int cunit_begin, int cunit_end,//cunit_end is the root of cunit
                          	 int *pattern_begin, int *pattern_length, int *pattern, //patterns
                          	 int *codes, int *types,// contexts
                          	 int *graph, int *oppo_table,
                          	 int *params_order,
                          	 short *truth_value_result,
                          	 int *link_result, int *link_num, int *cur_link_size,
                          	 int last_cunit_root,
                          	 int ccopy_num) {
	
	
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < ccopy_num) {

		Context params[MAX_PARAM_NUM];
		Context ordered_params[MAX_PARAM_NUM];

		for (int i = 0; i < MAX_PARAM_NUM; i++) {
            params[i].id = -1;
            ordered_params[i].id = -1;
         }

		int ccopy_root_offset = calc_offset(cunit_end, tid, params,
											parent, left_child, right_child, node_type, pattern_idx,
											pattern_begin, pattern_length, pattern,
											codes, types,
											branch_size);

//#ifdef DEBUG
//		printf("root = %d, ccopynum = %d, offset = %d\n",cunit_end, ccopy_num, ccopy_root_offset);
//#endif
		for (int node = cunit_begin; node <= cunit_end; node++) {
			int offset = ccopy_root_offset - (cunit_end - node);
			int type = node_type[node];
			bool value;

			Node* cur_links = &links[offset];
			init_node(cur_links);

			switch(type) {
				case UNIVERSAL_NODE: {
					int step = branch_size[left_child[node]];
					value = true;
					bool first = true;
					for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
						value = value && truth_values[offset - (i * step + 1)];
						if(!truth_values[offset - (i * step + 1)]) {
							if(first) {
								init_node(cur_links);
								first = false;
							}
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
					}

					break;
				}

				case EXISTENTIAL_NODE: {
					int step = branch_size[left_child[node]];
					value = false;
					bool first = true;
					for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
						value = value || truth_values[offset - (i * step + 1)];
						if(truth_values[offset - (i * step + 1)]) {
							if(first) {
								init_node(cur_links);
								first = false;
							}
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
					}
					break;
				}

				case AND_NODE: {
					//right && left
					value = truth_values[offset - 1] && truth_values[offset - (branch_size[right_child[node]] + 1)];

					if (truth_values[offset - 1] == value) {
						linkHelper(cur_links, &(links[offset - 1]));
					}

					if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
						linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}

					break;
				}
				case OR_NODE: {
					//right || left
					value = truth_values[offset - 1] || truth_values[offset - (branch_size[right_child[node]] + 1)];

					if (truth_values[offset - 1] == value) {
						linkHelper(cur_links, &(links[offset - 1]));
					}

					if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
						linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}

					break;
				}

				case IMPLIES_NODE: {
					//!left || right
					value = !truth_values[offset - (branch_size[right_child[node]] + 1)] || truth_values[offset - 1];

					if(value) {
	                   linkHelper(cur_links, &(links[offset - 1]));
	                   linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}
					else {
					   linkHelper(cur_links, &(links[offset - 1]));
					}

					break;
				}

				case NOT_NODE: {
					value = !truth_values[offset - 1];
					linkHelper(cur_links, &(links[offset - 1]));
					break;
				}

				default : { //BFUNC
					switch(type) {
						case BEFORE: {
							reorder_params(oppo_table, params_order, params, ordered_params);
							value = BEFORE(ordered_params, params_order[0]);
							break;
						}

						case GATE: {
							int *gate_params_order = params_order + (GATE - BEFORE) * (MAX_PATTERN_SIZE + 2);
							reorder_params(oppo_table, gate_params_order, params, ordered_params);
							value = gate(ordered_params[0], ordered_params[1]);
							break;
						}

						case EQUAL: {
							int *equal_params_order = params_order + (EQUAL - BEFORE) * (MAX_PATTERN_SIZE + 2);
							reorder_params(oppo_table, equal_params_order, params, ordered_params);
							value = equal(ordered_params[0], ordered_params[1]);
							break;
						}

						case CONN: {
							int *conn_params_order = params_order + (CONN - BEFORE) * (MAX_PATTERN_SIZE + 2);
							reorder_params(oppo_table, conn_params_order, params, ordered_params);
							value = conn(graph, params[0], params[1], conn_params_order[conn_params_order[0]]);
							break;
						}

						case OPPO: {
							int *oppo_params_order = params_order + (OPPO - BEFORE) * (MAX_PATTERN_SIZE + 2);
							reorder_params(oppo_table, oppo_params_order, params, ordered_params);
							value = oppo(oppo_table, ordered_params[0], ordered_params[1]);
							break;
						}

						case NEXT: {
						    int *next_params_order = params_order + (NEXT - BEFORE) * (MAX_PATTERN_SIZE + 2);
						    reorder_params(oppo_table, next_params_order, params, ordered_params);
						    value = next(ordered_params[0], ordered_params[1]);
						    break;
						}

					}

			
					for (int i = 0; i < MAX_PARAM_NUM; i++) {
						cur_links->params[i] = params[i].id;
					}
					break;
				}

				
			}

			truth_values[offset] = value;
		}

		if (last_cunit_root == cunit_end ) {
		    *truth_value_result = truth_values[ccopy_root_offset];
		    if(!truth_values[ccopy_root_offset]) {
            
         		int len = 0;
                for(Node *head = &links[ccopy_root_offset]; head != NULL; head = head ->next) {
                
                	if(len < MAX_LINK_SIZE) {
	                	for(int j = 0; j < MAX_PARAM_NUM; j++) {
	                         link_result[MAX_PARAM_NUM * len + j] = head->params[j];
	                    }
                	}

                    len++;
                }
                
                *cur_link_size = len;
                *link_num = len > MAX_LINK_SIZE ? MAX_LINK_SIZE : len;
         	}
        }
	}

 }

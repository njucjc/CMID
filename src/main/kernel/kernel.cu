#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

enum Type {
	NOT_NODE = 0,
	AND_NODE,
	IMPLIES_NODE,
	UNIVERSAL_NODE,
	EXISTENTIAL_NODE,
	BFUNC_NODE,
	EMPTY_NODE,
	SAME,
	SZ_SPD_CLOSE,
	SZ_LOC_CLOSE,
	SZ_LOC_DIST,
	SZ_LOC_DIST_NEQ ,
	SZ_LOC_RANGE,
	OR_NODE
};

#define MAX_PARAM_NUM 2
#define MAX_CCT_SIZE 3000000
#define MAX_LINK_SIZE 500
#define DEBUG

struct Context{
	int id;
	double latitude;
	double longitude;
	double speed;
	int plateNumber;
};

struct Node {
	Node *next;
	int params[MAX_PARAM_NUM];
};


__device__ bool truth_values[MAX_CCT_SIZE];
__device__ Node links[MAX_CCT_SIZE];

extern "C"
__device__ bool same(Context c1, Context c2){
	return (c1.plateNumber == c2.plateNumber);
}

extern "C"
__device__ bool sz_spd_close(Context c1, Context c2){
	return ((c1.speed - c2.speed) >= -50.0 && (c1.speed - c2.speed) <= 50.0);
}

extern "C"
__device__ bool sz_loc_close(Context c1, Context c2){
	return ((c1.latitude - c2.latitude) * (c1.latitude - c2.latitude) + (c1.longitude - c2.longitude) * (c1.longitude - c2.longitude)) <= 0.000001;
}

extern "C"
__device__ bool sz_loc_dist(Context c1, Context c2){
	return ((c1.latitude - c2.latitude) * (c1.latitude - c2.latitude) + (c1.longitude - c2.longitude) * (c1.longitude - c2.longitude)) <= 0.000625;
}

extern "C"
__device__ bool sz_loc_dist_neq(Context c1, Context c2){
	double dist = ((c1.latitude - c2.latitude) * (c1.latitude - c2.latitude) + (c1.longitude - c2.longitude) * (c1.longitude - c2.longitude));
	bool result = true;
    if (dist > 0.000625 || dist == 0) {
    	result = false;
    }
    return result;
	//return (dist <= 0.000625) && (dist != 0);
}

extern "C"
__device__ bool sz_loc_range(Context c){
	return c.longitude >= 112.0 && c.longitude <= 116.0 && c.latitude >=20.0 && c.latitude <= 24.0;
}

extern "C"
__device__ void linkHelper(Node *link1, Node *link2) {
	//inital and assumpt that link1 != null;

	if (link2 == NULL) {
		return;
	}

	Node *tail = link1;
	int len = 1;
	while(tail->next != NULL){
		tail = tail->next;
		len++;
	}

	for(Node *cur = link2; cur != NULL; ) {
		Node *p = link1;
		int i;
		for(i = 0; i < len; i++) {
			if(p->params[0] == cur->params[0] && p->params[1] == cur->params[1]) {
				break;
			}
			else {
				p = p->next;
			}
		}

		if(i == len) {
			q = cur;
			cur = cur->next;

			tail->next = q;
			q->next = NULL;
			tail = q;
		}
		else {
			cur = cur->next;
		}

	}
}

extern "C"
__device__ int calc_offset(	int node, int tid, Context *params,
							int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx,
							int *pattern_begin, int *pattern_length, int *pattern,
							double *longitude, double *latitude, double *speed, int *plateNumber, // contexts
							int *branch_size) {

	int offset = branch_size[node];
	int current_node = node;
	int index = 0, tmp = tid;
	while (parent[current_node] != -1) {
		int type = node_type[parent[current_node]];
		if (type == Type::EXISTENTIAL_NODE || type == Type::UNIVERSAL_NODE) {
			int len = pattern_length[pattern_idx[parent[current_node]]];
			int branch_idx = tmp % len;
			tmp /= len;

			params[index].id = pattern[pattern_begin[pattern_idx[parent[current_node]]] + branch_idx];//(pattern + pattern_idx[parent[current_node]] * MAX_PATTERN_SIZE)[(branch_idx + pattern_begin[pattern_idx[parent[current_node]]]) % MAX_PATTERN_SIZE];
			params[index].latitude = latitude[params[index].id];
			params[index].longitude = longitude[params[index].id];
			params[index].speed = speed[params[index].id];
			params[index].plateNumber = plateNumber[params[index].id];

			offset += branch_idx * branch_size[current_node] ;
//			printf("branch_idx = %d, branch_size = %d\n", branch_idx, branch_size[current_node]);
			index++;
		}
		else if (type == Type::AND_NODE || type == Type::IMPLIES_NODE || type == Type::OR_NODE) {
			if (right_child[parent[current_node]] == current_node) {
				offset += branch_size[left_child[parent[current_node]]];
			}
		}
		else {
		    offset += 0;
		}
		current_node = parent[current_node];
	}
	return offset - 1;
}

extern "C"
__global__ void evaluation(int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx, //constraint rule
                          	 int *branch_size, int cunit_begin, int cunit_end,//cunit_end is the root of cunit
                          	 int *pattern_begin, int *pattern_length, int *pattern, //patterns
                          	 double *longitude, double *latitude, double *speed,int *plateNumber,// contexts
                          	 short *truth_value_result,
                          	 int *link_result, int *link_num, int *cur_link_size,
                          	 int last_cunit_root,
                          	 int ccopy_num) {
	
	
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < ccopy_num) {

		Context params[MAX_PARAM_NUM];
		for (int i = 0; i < MAX_PARAM_NUM; i++) {
            params[i].id = -1;
         }
		int ccopy_root_offset = calc_offset(cunit_end, tid, params,
											parent, left_child, right_child, node_type, pattern_idx,
											pattern_begin, pattern_length, pattern,
											longitude, latitude, speed, plateNumber,
											branch_size);

//#ifdef DEBUG
//		printf("root = %d, ccopynum = %d, offset = %d\n",cunit_end, ccopy_num, ccopy_root_offset);
//#endif
		for (int node = cunit_begin; node <= cunit_end; node++) {
			int offset = ccopy_root_offset - (cunit_end - node);
			int type = node_type[node];
			bool value;

			Node* cur_links = &links[offset];
			cur_links->next = NULL;

			switch(type) {
				case Type::UNIVERSAL_NODE: {
					int step = branch_size[left_child[node]];
					value = true;
					bool first = true;
					for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
						value = value && truth_values[offset - (i * step + 1)];
						if(!truth_values[offset - (i * step + 1)]) {
							if(first) {
								cur_links->next = NULL;
								first = false;
							}
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
						else if(value) {
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
					}

					break;
				}

				case Type::EXISTENTIAL_NODE: {
					int step = branch_size[left_child[node]];
					value = false;
					bool first = true;
					for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
						value = value || truth_values[offset - (i * step + 1)];
						if(truth_values[offset - (i * step + 1)]) {
							if(first) {
								cur_links->next= NULL;
								first = false;
							}
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
						else if(!value) {
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
					}
					break;
				}

				case Type::AND_NODE: {
					//right && left
					value = truth_values[offset - 1] && truth_values[offset - (branch_size[right_child[node]] + 1)];

					if (truth_values[offset - 1] == value) {
						linkHelper(cur_links, &(links[offset - 1]));
					}

					if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
						linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}

					break;
				}
				case Type::OR_NODE: {
					//right || left
					value = truth_values[offset - 1] || truth_values[offset - (branch_size[right_child[node]] + 1)];

					if (truth_values[offset - 1] == value) {
						linkHelper(cur_links, &(links[offset - 1]));
					}

					if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
						linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}

					break;
				}

				case Type::IMPLIES_NODE: {
					//!left || right
					value = !truth_values[offset - (branch_size[right_child[node]] + 1)] || truth_values[offset - 1];

					if(value) {
	                   linkHelper(cur_links, &(links[offset - 1]));
	                   linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}
					else {
					   linkHelper(cur_links, &(links[offset - 1]));
					}

					break;
				}

				case Type::NOT_NODE: {
					value = !truth_values[offset - 1];
					linkHelper(cur_links, &(links[offset - 1]));
					break;
				}

				default : { //BFUNC
					switch(type) {
						case Type::SAME: {
							value = same(params[0], params[1]);
							break;
						}

						case Type::SZ_SPD_CLOSE: {
							value = sz_spd_close(params[0], params[1]);
							break;
						}

						case Type::SZ_LOC_CLOSE: {
							value = sz_loc_close(params[0], params[1]);
							break;
						}

						case Type::SZ_LOC_DIST: {
							value = sz_loc_dist(params[0], params[1]);
							break;
						}

						case Type::SZ_LOC_DIST_NEQ: {
							value = sz_loc_dist_neq(params[0], params[1]);
							break;
						}

						case Type::SZ_LOC_RANGE: {
							value = sz_loc_range(params[0]);
							break;
						}
					}

					cur_links->next = NULL;
					for (int i = 0; i < MAX_PARAM_NUM; i++) {
						cur_links->params[i] = params[i].id;
					}
					break;
				}

				
			}

			truth_values[offset] = value;
		}

		if (last_cunit_root == cunit_end ) {
		    *truth_value_result = truth_values[ccopy_root_offset];
		    if(!truth_values[ccopy_root_offset]) {
            
         		int len = 0;
                for(Node *head = &links[ccopy_root_offset]; head != NULL; head = head ->next) {
                
                	if(len < MAX_LINK_SIZE) {
	                	for(int j = 0; j < MAX_PARAM_NUM; j++) {
	                         link_result[MAX_PARAM_NUM * len + j] = links[ccopy_root_offset].params[j];
	                    }
                	}

                    len++;
                }
                
                *cur_link_size = len;
                *link_num = len > MAX_LINK_SIZE ? MAX_LINK_SIZE : len;
         	}
        }
	}

 }